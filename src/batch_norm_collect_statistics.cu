#include "hip/hip_runtime.h"
#include "../include/batch_norm_collect_statistics.cuh"

__global__ void batch_norm_collect_statistics_kernel(
    const float* __restrict__ input, int height, int width, int depth,
    float epsilon, float* __restrict__ save_mean,
    float* __restrict__ save_transformed_var) {
    __shared__ int shared_n[2 * 2 * C10_WARP_SIZE +
                            C10_WARP_SIZE];  // Shared memory for storing
                                             // intermediate results
    __shared__ float
        shared_avg_var[2 * C10_WARP_SIZE];  // Storage for averages and
                                            // variances in shared memory

    int plane = blockIdx.x;
    int N = height * depth;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;

    // Compute the mean and variance across (batch, x/y/z)
    float avg = 0;
    float var_n = 0;
    int n = 0;
    for (int batch = threadIdx.y; batch < height; batch += blockDim.y) {
        for (int x = threadIdx.x; x < depth; x += blockDim.x) {
            float v = input[batch * width * depth + plane * depth + x];
            float d1 = v - avg;
            n++;
            avg += d1 / n;
            var_n += d1 * (v - avg);
        }
    }

    // Parallel reduction in warp
    for (int i = 0; i < getMSB(C10_WARP_SIZE); ++i) {
        float o_avg = WARP_SHFL_XOR(avg, 1 << i, C10_WARP_SIZE);
        int o_n = WARP_SHFL_XOR(n, 1 << i, C10_WARP_SIZE);
        float factor = 1.0 / fmaxf(1.0, n + o_n);
        var_n += WARP_SHFL_XOR(var_n, 1 << i, C10_WARP_SIZE) +
                 (avg - o_avg) * (avg - o_avg) * n * o_n * factor;
        avg = (n * avg + o_n * o_avg) * factor;
        n += o_n;
    }

    // Write each warp's result into shared memory
    __syncthreads();
    if (tid % C10_WARP_SIZE == 0) {
        shared_n[tid / C10_WARP_SIZE] = n;
        shared_avg_var[tid / C10_WARP_SIZE * 2] = avg;
        shared_avg_var[tid / C10_WARP_SIZE * 2 + 1] = var_n;
    }
    __syncthreads();

    // Final reduction from shared memory to a single number per block
    if (tid < C10_WARP_SIZE) {
        n = (tid < blockDim.x * blockDim.y / C10_WARP_SIZE ? shared_n[tid] : 0);
        avg = (tid < blockDim.x * blockDim.y / C10_WARP_SIZE
                   ? shared_avg_var[2 * tid]
                   : 0);
        var_n = (tid < blockDim.x * blockDim.y / C10_WARP_SIZE
                     ? shared_avg_var[2 * tid + 1]
                     : 0);
    }
    for (int i = 0; i < getMSB(C10_WARP_SIZE); ++i) {
        float o_avg = WARP_SHFL_XOR(avg, 1 << i, C10_WARP_SIZE);
        int o_n = WARP_SHFL_XOR(n, 1 << i, C10_WARP_SIZE);
        float factor = 1.0 / fmaxf(1.0, n + o_n);
        var_n += WARP_SHFL_XOR(var_n, 1 << i, C10_WARP_SIZE) +
                 (avg - o_avg) * (avg - o_avg) * n * o_n * factor;
        avg = (n * avg + o_n * o_avg) * factor;
        n += o_n;
    }

    // Save the mean and variance, using transformations as needed
    if (tid == 0) {
        if (save_mean != NULL) {
            save_mean[plane] = avg;
        }
        if (save_transformed_var != NULL) {
            save_transformed_var[plane] =
                var_n / N + epsilon;  // Assuming the transformation is to add
                                      // epsilon (modify as needed)
        }
    }
}

