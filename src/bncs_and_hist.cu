#include "../include/bncs_and_hist.cuh"

float benchmark_bncs_and_hist(const float* h_input, int height, int width,
                              int depth, float epsilon, float* h_mean,
                              float* h_transformed_var, float* h_a, /* output */
                              const float* h_b,                     /* input */
                              int nbins, float minvalue, float maxvalue,
                              int k2_totalElements, const int loop) {
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    // Init for bncs
    float *d_input, *d_mean, *d_transformed_var;
    const int total_elements = height * width * depth;
    hipMalloc(&d_input, total_elements * sizeof(float));
    hipMalloc(&d_mean, width * sizeof(float));
    hipMalloc(&d_transformed_var, width * sizeof(float));
    hipMemcpy(d_input, h_input, total_elements * sizeof(float),
               hipMemcpyHostToDevice);

    // Init for hist
    size_t k2_size = k2_totalElements * sizeof(float);
    float *d_a, *d_b;
    hipMalloc(&d_a, nbins * sizeof(float));
    hipMalloc(&d_b, k2_size);
    hipMemcpy(d_b, h_b, k2_size, hipMemcpyHostToDevice);
    hipMemset(d_a, 0, nbins * sizeof(float));

    // Configure bncs kernel
    dim3 blocks(width);    // One block per channel
    dim3 threads(16, 64);  // 1024 threads per block

    // Configure hist kernel
    int threadsPerBlock = 128;
    int blocksPerGrid = 128;

    // Warm up
    for (int i = 0; i < 5; i++) {
        batch_norm_collect_statistics_kernel<<<blocks, threads, 0, stream0>>>(
            d_input, height, width, depth, epsilon, d_mean, d_transformed_var);
        histogram1D_kernel<<<blocksPerGrid, threadsPerBlock,
                             nbins * sizeof(float), stream1>>>(
            d_a, d_b, nbins, minvalue, maxvalue, k2_totalElements);
        hipDeviceSynchronize();
    }

    float msec = 0.0;
    float total = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int i = 0; i < loop; i++) {
        hipEventRecord(start);
        batch_norm_collect_statistics_kernel<<<blocks, threads, 0, stream0>>>(
            d_input, height, width, depth, epsilon, d_mean, d_transformed_var);
        histogram1D_kernel<<<blocksPerGrid, threadsPerBlock,
                             nbins * sizeof(float), stream1>>>(
            d_a, d_b, nbins, minvalue, maxvalue, k2_totalElements);
        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&msec, start, stop);
        total += msec;
    }

    // Copy results back to host
    hipMemcpy(h_mean, d_mean, width * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_transformed_var, d_transformed_var, width * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_a, d_a, nbins * sizeof(float), hipMemcpyDeviceToHost);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    // Free device memory
    hipFree(d_input);
    hipFree(d_mean);
    hipFree(d_transformed_var);
    hipFree(d_a);
    hipFree(d_b);
    return total / loop;
}