#include "../include/gemm_cublas.h"

/*
 * This function refers to NVIDIA's example
 * (https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLAS/Level-3/gemm/cublas_gemm_example.cu)
 */
void gemm_cublas(float* input_a, float* input_b, float* input_c, const int M,
                 const int N, const int K) {
    const int lda = K;
    const int ldb = N;
    const int ldc = N;

    hipblasHandle_t cublasH = nullptr;
    hipStream_t stream = nullptr;
    hipblasCreate(&cublasH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasSetStream(cublasH, stream);

    const float alpha = 1.0;
    const float beta = 1.0;

    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c = nullptr;

    hipMalloc(reinterpret_cast<void**>(&d_a), sizeof(float) * M * K);
    hipMalloc(reinterpret_cast<void**>(&d_b), sizeof(float) * K * N);
    hipMalloc(reinterpret_cast<void**>(&d_c), sizeof(float) * M * N);

    hipMemcpyAsync(d_a, input_a, sizeof(float) * M * K, hipMemcpyHostToDevice,
                    stream);
    hipMemcpyAsync(d_b, input_b, sizeof(float) * K * N, hipMemcpyHostToDevice,
                    stream);
    hipMemcpyAsync(d_c, input_c, sizeof(float) * M * N, hipMemcpyHostToDevice,
                    stream);

    // /*
    //  * Transposing problem in cuBLAS:
    //  * All matrices are considered in column-major in cuBLAS.
    //  * We want to do row-major gemm, so we need to do transpose.
    //  * We can do transpose for A and B with cublas api: transa and transb,
    //  * but transc is not offered.
    //  * How to handle it:
    //  * C = A * B => C^T = B^T * A^T
    //  * What we want is C^T, so we can switch the order of A and B.
    //  * And the A and B is row-major, they are A^T and B^T in cublas sight, so
    //  * transa and transb are all HIPBLAS_OP_N (no need for transposing).
    //  * In one sentence: if we want do "C(row-major) = A(row-major) *
    //  * B(row-major)", do "C = B * A" in cublas.
    //  */
    // hipblasOperation_t transa = HIPBLAS_OP_N;
    // hipblasOperation_t transb = HIPBLAS_OP_N;
    // hipblasSgemm(cublasH, transa, transb, M, N, K, &alpha, d_b, ldb, d_a, lda,
    //             &beta, d_c, ldc);

    /*
     * Now Matrix a is row-major and matrix b is col-major
     * We want C^T, and C^T = B^T * A^T => B * A^T (recall matrix b is
     * col-major)
     */
    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasSgemm(cublasH, transa, transb, M, N, K, &alpha, d_b, ldb, d_a, lda,
                &beta, d_c, ldc);

    hipMemcpyAsync(input_c, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost,
                    stream);

    hipStreamDestroy(stream);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

float bench_gemm_cublas(float* input_a, float* input_b, float* input_c,
                        const int M, const int N, const int K, const int loop) {
    const int lda = K;
    const int ldb = N;
    const int ldc = N;

    hipblasHandle_t cublasH = nullptr;
    hipStream_t stream = nullptr;
    hipblasCreate(&cublasH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasSetStream(cublasH, stream);

    const float alpha = 1.0;
    const float beta = 1.0;

    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c = nullptr;

    hipMalloc(reinterpret_cast<void**>(&d_a), sizeof(float) * M * K);
    hipMalloc(reinterpret_cast<void**>(&d_b), sizeof(float) * K * N);
    hipMalloc(reinterpret_cast<void**>(&d_c), sizeof(float) * M * N);

    hipMemcpyAsync(d_a, input_a, sizeof(float) * M * K, hipMemcpyHostToDevice,
                    stream);
    hipMemcpyAsync(d_b, input_b, sizeof(float) * K * N, hipMemcpyHostToDevice,
                    stream);
    hipMemcpyAsync(d_c, input_c, sizeof(float) * M * N, hipMemcpyHostToDevice,
                    stream);

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    // Warm up
    for (int i = 0; i < 5; i++) {
        hipblasSgemm(cublasH, transa, transb, M, N, K, &alpha, d_b, ldb, d_a,
                    lda, &beta, d_c, ldc);
    }

    float msec = 0.0;
    float total = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int i = 0; i < loop; i++) {
        hipEventRecord(start);
        hipblasSgemm(cublasH, transa, transb, M, N, K, &alpha, d_b, ldb, d_a,
                    lda, &beta, d_c, ldc);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&msec, start, stop);
        total += msec;
    }

    hipStreamDestroy(stream);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return total / loop;
}