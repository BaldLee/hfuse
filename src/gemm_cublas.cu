#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "../include/gemm_cublas.cuh"

/*
 * This function refers to NVIDIA's example
 * (https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLAS/Level-3/gemm/cublas_gemm_example.cu)
 */
void gemm_cublas(float* input_a, float* input_b, float* input_c, const int M,
                 const int N, const int K) {
    const int lda = K;
    const int ldb = N;
    const int ldc = N;

    hipblasHandle_t cublasH = nullptr;
    hipStream_t stream = nullptr;
    hipblasCreate(&cublasH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasSetStream(cublasH, stream);

    const float alpha = 1.0;
    const float beta = 0.0;

    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c = nullptr;

    hipMalloc(reinterpret_cast<void**>(&d_a), sizeof(float) * M * K);
    hipMalloc(reinterpret_cast<void**>(&d_b), sizeof(float) * K * N);
    hipMalloc(reinterpret_cast<void**>(&d_c), sizeof(float) * M * N);

    hipMemcpyAsync(d_a, input_a, sizeof(float) * M * K, hipMemcpyHostToDevice,
                    stream);
    hipMemcpyAsync(d_b, input_b, sizeof(float) * K * N, hipMemcpyHostToDevice,
                    stream);
    hipMemcpyAsync(d_c, input_c, sizeof(float) * M * N, hipMemcpyHostToDevice,
                    stream);

    /*
     * Transposing problem in cuBLAS:
     * All matrices are considered in column-major in cuBLAS.
     * We want to do row-major gemm, so we need to do transpose.
     * We can do transpose for A and B with cublas api: transa and transb,
     * but transc is not offered.
     * How to handle it:
     * C = A * B => C^T = B^T * A^T
     * What we want is C^T, so we can switch the order of A and B.
     * And the A and B is row-major, they are A^T and B^T in cublas sight, so
     * transa and transb are all HIPBLAS_OP_N (no need for transposing).
     * In one sentence: if we want do "C(row-major) = A(row-major) * B(row-major)",
     * do "C = B * A" in cublas.
     */
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasSgemm(cublasH, transa, transb, M, N, K, &alpha, d_b, ldb, d_a, lda,
                &beta, d_c, ldc);

    hipMemcpyAsync(input_c, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost,
                    stream);
}