#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../include/histogram1d_gpu.cuh"

/* This kernel comes from pytorch/aten/src/ATen/native/cuda/SummaryOps.cu
 * (pytorch/pytorch commit d59f1da6)
 *
 * The PART A,B,C is devided according to the paper: Automatic Horizontal Fusion
 * for GPU Kernels (https://dblp.org/rec/conf/cgo/LiZPL22)
 *
 * We assume the "getop" is "getDummyOp", which always returns 1. In origin
 * code, "getop" is used to get the weight from weight tensor c. The tensor info
 * of c is passed by the anonymous function "getWeightsOp". Refer to
 * CUDA_tensor_histogram in pytorch/aten/src/ATen/native/cuda/SummaryOps.cu for
 * details.
 */
__global__ void histogram1D_kernel(float* a,       /* output */
                                   const float* b, /* input */
                                   int nbins, float minvalue, float maxvalue,
                                   int totalElements) {
    extern __shared__ float smem[];
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // PARTA:Initialize shared memory counters
    for (int idx = threadIdx.x; idx < nbins; idx += blockDim.x) {
        smem[idx] = 0;
    }
    __syncthreads();

    // PART B: Go over the input b to increment shared counters
    while (i < totalElements) {
        float bVal = b[i];
        if (bVal >= minvalue && bVal <= maxvalue) {
            int bin = static_cast<int>((bVal - minvalue) /
                                       (maxvalue - minvalue) * nbins);
            atomicAdd(&smem[bin], 1);
        }
        i += blockDim.x * gridDim.x;
    }
    __syncthreads();

    // PART C: Increment the output a with the shared counters
    for (int idx = threadIdx.x; idx < nbins; idx += blockDim.x) {
        atomicAdd(&a[idx], smem[idx]);
    }
}

void histogram1D_gpu(float* h_a,       /* output */
                     const float* h_b, /* input */
                     int nbins, float minvalue, float maxvalue,
                     int totalElements) {
    size_t size = totalElements * sizeof(float);

    // Allocate memory
    float *d_a, *d_b;
    hipMalloc(&d_a, nbins * sizeof(float));
    hipMalloc(&d_b, size);

    // Copy data to device
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipMemset(d_a, 0, nbins * sizeof(float));

    // Configure kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (totalElements + threadsPerBlock - 1) / threadsPerBlock;

    histogram1D_kernel<<<blocksPerGrid, threadsPerBlock,
                         nbins * sizeof(float)>>>(d_a, d_b, nbins, minvalue,
                                                  maxvalue, totalElements);

    hipMemcpy(h_a, d_a, nbins * sizeof(float), hipMemcpyDeviceToHost);
}